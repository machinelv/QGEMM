#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>

#include "gpu_lib.h"
#include "gpu_types.h"
#include "fast_quat.h"


namespace gemm_kernel {
namespace normal_gemm {
namespace sm80 {

template<
    size_t BLOCK_M, size_t BLOCK_N, size_t BLOCK_K,
    size_t GROUP_M, size_t STAGE >
struct GEMM_Config {
    size_t BLOCK_TILE_SIZE_M = BLOCK_M;   
    size_t BLOCK_TILE_SIZE_N = BLOCK_N;
    size_t BLOCK_TILE_SIZE_K = BLOCK_K;
    size_t GROUP_SIZE_M      = GROUP_M;
    size_t STAGE_NUMS        = STAGE;
};
struct WMMA_Config {
    static constexpr size_t WARP_TILE_SIZE_M  = 32;    // wmma number of rows in a warp
    static constexpr size_t WARP_TILE_SIZE_N  = 32;    // wmma number of columns in a warp
    static constexpr size_t WMMA_TILE_SIZE_M   = 16;
    static constexpr size_t WMMA_TILE_SIZE_N   = 16;
    static constexpr size_t WMMA_TILE_SIZE_K   = 32;
}


constexpr GEMM_Config<256, 256, 64, 2, 0> gemmconf_256x256x64;
constexpr GEMM_Config<128, 128, 64, 2, 0> gemmconf_128x128x64;
constexpr GEMM_Config<64, 128, 64, 2, 0> gemmconf_64x128x64;
constexpr GEMM_Config<64, 64, 64, 2, 0> gemmconf_64x64x64;




template <typename T, typename T_OUTPUT,
            size_t BLOCK_TILE_SIZE_M, size_t BLOCK_TILE_SIZE_N, size_t BLOCK_TILE_SIZE_K,
            size_t WARP_TILE_SIZE_M, size_t WARP_TILE_SIZE_N, 
            size_t WMMA_TILE_PER_WARP_M, size_t WMMA_TILE_PER_WARP_N, size_t WMMA_TILE_PER_WARP_K,
            size_t THREAD_TILE_SIZE_M_SCALE, size_t THREAD_TILE_SIZE_N_SCALE,
            size_t WMMA_TILE_SIZE_M, size_t WMMA_TILE_SIZE_N, size_t WMMA_TILE_SIZE_K>
inline __device__ void process_data_from_shared_memory_using_wmma(
        const T A_T_shared_block_tile[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_M],
        const T B_shared_block_tile[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_N],
        float32_t A_scale_thread_tile[THREAD_TILE_SIZE_M_SCALE], float32_t B_scale_thread_tile[THREAD_TILE_SIZE_N_SCALE],
        wmma::fragment<matrix_a, WMMA_TILE_SIZE_M, WMMA_TILE_SIZE_N, WMMA_TILE_SIZE_K, T, col_major> 
        a_frag[WMMA_TILE_PER_WARP_M],
        wmma::fragment<matrix_b, WMMA_TILE_SIZE_M, WMMA_TILE_SIZE_N, WMMA_TILE_SIZE_K, T, row_major>
        b_frag[WMMA_TILE_PER_WARP_N],
        wmma::fragment<accumulator, WMMA_TILE_SIZE_M, WMMA_TILE_SIZE_N, WMMA_TILE_SIZE_K, float32_t> 
        acc_frag_fp32[WMMA_TILE_PER_WARP_M][WMMA_TILE_PER_WARP_N],
        size_t M, size_t N, size_t K,
        size_t warp_m_id, size_t warp_n_id
    )
{
    wmma::fragment<accumulator, WMMA_TILE_SIZE_M, WMMA_TILE_SIZE_N, WMMA_TILE_SIZE_K, float32_t> acc_frag_fp32_local[WMMA_TILE_PER_WARP_M][WMMA_TILE_PER_WARP_N];
    #pragma clang loop unroll_count(WMMA_TILE_PER_WARP_M)
    for (size_t wmma_tile_m_idx{0U}; wmma_tile_m_idx < WMMA_TILE_PER_WARP_M; ++wmma_tile_m_idx){
        #pragma clang loop unroll_count(WMMA_TILE_PER_WARP_N)
        for (size_t wmma_tile_n_idx{0U}; wmma_tile_n_idx < WMMA_TILE_PER_WARP_N;++wmma_tile_n_idx) {
            wmma::fill_fragment(acc_frag_fp32_local[wmma_tile_m_idx][wmma_tile_n_idx], static_cast<float32_t>(0));
        }
    }

    // process wmma tile
    #pragma clang loop unroll_count(WMMA_TILE_PER_WARP_K)
    for (size_t wmma_tile_idx_k{0U}; wmma_tile_idx_k < WMMA_TILE_PER_WARP_K; ++wmma_tile_idx_k) {
        // Load data from shared memory to register
        size_t block_tile_wmma_tile_k_idx{wmma_tile_idx_k * WMMA_TILE_SIZE_K};
        // Load A and B matrices from shared memory to registers
        #pragma clang loop unroll_count(WMMA_TILE_PER_WARP_M)
        for (size_t wmma_tile_idx_m{0U}; wmma_tile_idx_m < WMMA_TILE_PER_WARP_M; ++wmma_tile_idx_m) {
            size_t block_tile_wmma_tile_m_idx{warp_m_id * WARP_TILE_SIZE_M + wmma_tile_idx_m * WMMA_TILE_SIZE_M};
            wmma::load_matrix_sync(a_frag[wmma_tile_idx_m], &A_T_shared_block_tile[block_tile_wmma_tile_k_idx][block_tile_wmma_tile_m_idx], BLOCK_TILE_SIZE_M);
        }
        #pragma clang loop unroll_count(WMMA_TILE_PER_WARP_N)
        for (size_t wmma_tile_idx_n{0U}; wmma_tile_idx_n < WMMA_TILE_PER_WARP_N; ++wmma_tile_idx_n) {
            size_t block_tile_wmma_tile_n_idx{warp_n_id * WARP_TILE_SIZE_N + wmma_tile_idx_n * WMMA_TILE_SIZE_N};
            wmma::load_matrix_sync(b_frag[wmma_tile_idx_n], &B_shared_block_tile[block_tile_wmma_tile_k_idx][block_tile_wmma_tile_n_idx], BLOCK_TILE_SIZE_N);
        }

        // Compute the acc_frag
        #pragma clang loop unroll_count(WMMA_TILE_PER_WARP_M)
        for (size_t wmma_tile_idx_m{0U}; wmma_tile_idx_m < WMMA_TILE_PER_WARP_M; ++wmma_tile_idx_m) {
            #pragma clang loop unroll_count(WMMA_TILE_PER_WARP_N)
            for (size_t wmma_tile_idx_n{0U}; wmma_tile_idx_n < WMMA_TILE_PER_WARP_N; ++wmma_tile_idx_n) {
                wmma::mma_sync(acc_frag_fp32_local[wmma_tile_idx_m][wmma_tile_idx_n], a_frag[wmma_tile_idx_m], b_frag[wmma_tile_idx_n], acc_frag_fp32_local[wmma_tile_idx_m][wmma_tile_idx_n]);
            }
        }
    }
    __syncthreads();

    // Scale the acc_frag_fp32 using A_scale and B_scale
    #pragma clang loop unroll_count(WMMA_TILE_PER_WARP_M)
    for (size_t wmma_tile_idx_m{0U}; wmma_tile_idx_m < WMMA_TILE_PER_WARP_M; ++wmma_tile_idx_m) {
        size_t acc_group_start = wmma_tile_idx_m * FP8_ACC_REG_REG_NUMBER;
        #pragma clang loop unroll_count(WMMA_TILE_PER_WARP_N)
        for (size_t wmma_tile_idx_n{0U}; wmma_tile_idx_n < WMMA_TILE_PER_WARP_N; ++wmma_tile_idx_n) {
            // acc_frag_fp32[wmma_tile_idx_m][wmma_tile_idx_n] *= A_scale_thread_tile[0] * B_scale_thread_tile[0];
            // 16 x 16 = 256 acc registers
            // 4 registers per thread
            #pragma clang loop unroll_count(FP8_ACC_REG_REG_NUMBER)
            for (size_t reg_idx{0U}; reg_idx < FP8_ACC_REG_REG_NUMBER; ++reg_idx) {
                acc_frag_fp32[wmma_tile_idx_m][wmma_tile_idx_n].x[reg_idx] += acc_frag_fp32_local[wmma_tile_idx_m][wmma_tile_idx_n].x[reg_idx] * A_scale_thread_tile[acc_group_start + reg_idx] * B_scale_thread_tile[0];
            }
        }
    }
}



template <typename T_INPUT, typename T_OUTPUT, 
            size_t BLOCK_TILE_SIZE_M, size_t BLOCK_TILE_SIZE_N, size_t BLOCK_TILE_SIZE_K,
            size_t WARP_TILE_SIZE_M, size_t WARP_TILE_SIZE_N, 
            size_t WMMA_TILE_SIZE_M, size_t WMMA_TILE_SIZE_N, size_t WMMA_TILE_SIZE_K,
            size_t GROUP_SIZE_M>
__global__ void FP8_GEMM_kernel(const T_INPUT* A, const T_INPUT* B, 
                    const float* A_scale, const float* B_scale, 
                    T_OUTPUT* C, 
                    const size_t M, const size_t N, const size_t K
                )
{
    static_assert(BLOCK_TILE_SIZE_K <= gemmconf::SCALE_BLOCK_SIZE, 
                  "BLOCK_TILE_SIZE_K must be less than or equal to SCALE_BLOCK_SIZE");
    static_assert(BLOCK_TILE_SIZE_M % WARP_TILE_SIZE_M == 0, 
                  "BLOCK_TILE_SIZE_M must be divisible by WARP_TILE_SIZE_M");
    static_assert(BLOCK_TILE_SIZE_N % WARP_TILE_SIZE_N == 0,
                    "BLOCK_TILE_SIZE_N must be divisible by WARP_TILE_SIZE_N");
    static_assert(BLOCK_TILE_SIZE_K % WMMA_TILE_SIZE_K == 0,   
                    "BLOCK_TILE_SIZE_M must be divisible by WMMA_TILE_SIZE_M");

    constexpr size_t WARP_NUM_M{BLOCK_TILE_SIZE_M / WARP_TILE_SIZE_M};
    constexpr size_t WARP_NUM_N{BLOCK_TILE_SIZE_N / WARP_TILE_SIZE_N};
    constexpr size_t WARP_NUM{WARP_NUM_M * WARP_NUM_N};
    constexpr size_t THREAD_NUM{WARP_NUM * WARP_SIZE};

    constexpr size_t WMMA_TILE_PER_WARP_M{WARP_TILE_SIZE_M / WMMA_TILE_SIZE_M};
    constexpr size_t WMMA_TILE_PER_WARP_N{WARP_TILE_SIZE_N / WMMA_TILE_SIZE_N};
    constexpr size_t WMMA_TILE_PER_WARP_K{BLOCK_TILE_SIZE_K / WMMA_TILE_SIZE_K};

    // Calculate the block row and block column indices
    const size_t block_id = blockIdx.x + blockIdx.y * gridDim.x;
    const size_t thread_id = threadIdx.x + threadIdx.y * blockDim.x;
    const size_t warp_id = thread_id / WARP_SIZE;
    const size_t warp_m_id = warp_id / WARP_NUM_N; // warp_row_idx
    const size_t warp_n_id = warp_id % WARP_NUM_N; // warp_col_idx

    size_t block_tile_num_m = (M + BLOCK_TILE_SIZE_M - 1) / BLOCK_TILE_SIZE_M;
    size_t block_tile_num_n = (N + BLOCK_TILE_SIZE_N - 1) / BLOCK_TILE_SIZE_N;

    size_t block_tile_start_m, block_tile_start_n;

    if constexpr (GROUP_SIZE_M > 1) {
        size_t group_block_num = GROUP_SIZE_M * block_tile_num_n;
        size_t group_id = block_id / group_block_num;
        size_t group_block_start_m = group_id * GROUP_SIZE_M;
        size_t group_block_size_m = min(GROUP_SIZE_M, M - group_block_start_m);

        size_t block_tile_id_m = group_block_start_m + ((block_id % group_block_num) % group_block_size_m);
        size_t block_tile_id_n = (block_id % group_block_num) / group_block_size_m;
        block_tile_start_m = block_tile_id_m * BLOCK_TILE_SIZE_M;
        block_tile_start_n = block_tile_id_n * BLOCK_TILE_SIZE_N;
    } else {
        block_tile_start_m = blockIdx.y * BLOCK_TILE_SIZE_M;
        block_tile_start_n = blockIdx.x * BLOCK_TILE_SIZE_N;
    }


    __shared__ T_INPUT A_T_block_tile[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_M];
    __shared__ T_INPUT B_block_tile[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_N];

    constexpr size_t THREAD_TILE_SIZE_K_SCALE = 1; // ((BLOCK_TILE_SIZE_K + gemmconf::SCALE_BLOCK_SIZE - 1) / gemmconf::SCALE_BLOCK_SIZE);
    constexpr size_t THREAD_TILE_SIZE_N_SCALE = 1; // ((BLOCK_TILE_SIZE_N + gemmconf::SCALE_BLOCK_SIZE - 1) / gemmconf::SCALE_BLOCK_SIZE);
    constexpr size_t THREAD_TILE_SIZE_M_SCALE = FP8_ACC_REG_REG_NUMBER * WMMA_TILE_PER_WARP_M; // (BLOCK_TILE_SIZE_M + THREAD_NUM - 1) / THREAD_NUM;
    float A_scale_thread_tile[THREAD_TILE_SIZE_M_SCALE];          // store in shared_memory, Note: Assuming that BLOCK_TILE_SIZE_K <= SCALE_BLOCK_SIZE
    float B_scale_thread_tile[THREAD_TILE_SIZE_N_SCALE];          // store in register
    
    wmma::fragment<accumulator, WMMA_TILE_SIZE_M, WMMA_TILE_SIZE_N, WMMA_TILE_SIZE_K, float32_t> acc_frag_fp32[WMMA_TILE_PER_WARP_M][WMMA_TILE_PER_WARP_N];
    #pragma clang loop unroll_count(WMMA_TILE_PER_WARP_M)
    for (size_t wmma_tile_m_idx{0U}; wmma_tile_m_idx < WMMA_TILE_PER_WARP_M; ++wmma_tile_m_idx){
        #pragma clang loop unroll_count(WMMA_TILE_PER_WARP_N)
        for (size_t wmma_tile_n_idx{0U}; wmma_tile_n_idx < WMMA_TILE_PER_WARP_N;++wmma_tile_n_idx) {
            wmma::fill_fragment(acc_frag_fp32[wmma_tile_m_idx][wmma_tile_n_idx], static_cast<float32_t>(0));
        }
    }
    
    for (size_t block_id_k{0}; block_id_k < K; block_id_k += BLOCK_TILE_SIZE_K)
    {  
        wmma::fragment<matrix_a, WMMA_TILE_SIZE_M, WMMA_TILE_SIZE_N, WMMA_TILE_SIZE_K, T_INPUT, col_major> a_frag[WMMA_TILE_PER_WARP_M];
        wmma::fragment<matrix_b, WMMA_TILE_SIZE_M, WMMA_TILE_SIZE_N, WMMA_TILE_SIZE_K, T_INPUT, row_major> b_frag[WMMA_TILE_PER_WARP_N];
        size_t block_tile_start_k = block_id_k;
        // load A and B matrices from global memory to shared memory
        load_data_from_global_memory_to_shared_memory_transposed_vectorized
            <T_INPUT, BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N, BLOCK_TILE_SIZE_K,
            WARP_TILE_SIZE_M, WARP_TILE_SIZE_N, THREAD_NUM>
            (A, B, A_T_block_tile, B_block_tile, M, N, K,
             block_tile_start_m, block_tile_start_n, block_tile_start_k, thread_id);
        // load A_scale and B_scale from global memory to registers
        load_data_from_global_memory_to_register<T_INPUT, WARP_TILE_SIZE_M, WARP_TILE_SIZE_N, WMMA_TILE_SIZE_M, THREAD_TILE_SIZE_M_SCALE, THREAD_TILE_SIZE_N_SCALE, THREAD_NUM>
            (A_scale, B_scale, A_scale_thread_tile, B_scale_thread_tile,
             M, N, K, block_tile_start_m, block_tile_start_n, block_tile_start_k, warp_m_id, warp_n_id, thread_id);
        __syncthreads();
        
        // load a_frag and b_frag from shared memory to registers and compute
        process_data_from_shared_memory_using_wmma<T_INPUT, T_OUTPUT, 
            BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N, BLOCK_TILE_SIZE_K,
            WARP_TILE_SIZE_M, WARP_TILE_SIZE_N, 
            WMMA_TILE_PER_WARP_M, WMMA_TILE_PER_WARP_N, WMMA_TILE_PER_WARP_K,
            THREAD_TILE_SIZE_M_SCALE, THREAD_TILE_SIZE_N_SCALE,
            WMMA_TILE_SIZE_M, WMMA_TILE_SIZE_N, WMMA_TILE_SIZE_K>
            (A_T_block_tile, B_block_tile, A_scale_thread_tile, B_scale_thread_tile,
                a_frag, b_frag, acc_frag_fp32, M, N, K, warp_m_id, warp_n_id);
        __syncthreads();
    }

    static_assert(THREAD_TILE_SIZE_M_SCALE == 8, "THREAD_TILE_SIZE_M_SCALE must be 8");
    static_assert(THREAD_TILE_SIZE_N_SCALE == 1, "THREAD_TILE_SIZE_N_SCALE must be 1");
    

    // Store the result to global memory
    #pragma clang loop unroll_count(WMMA_TILE_PER_WARP_M)
    for (size_t wmma_tile_idx_m{0U}; wmma_tile_idx_m < WMMA_TILE_PER_WARP_M; ++wmma_tile_idx_m) {
        #pragma clang loop unroll_count(WMMA_TILE_PER_WARP_N)
        for (size_t wmma_tile_idx_n{0U}; wmma_tile_idx_n < WMMA_TILE_PER_WARP_N; ++wmma_tile_idx_n) {
            wmma::fragment<accumulator, WMMA_TILE_SIZE_M, WMMA_TILE_SIZE_N, WMMA_TILE_SIZE_K, T_OUTPUT> acc_frag;
            #pragma clang loop unroll_count(FP8_ACC_REG_REG_NUMBER)
            for (size_t reg_idx{0U}; reg_idx < FP8_ACC_REG_REG_NUMBER; ++reg_idx) {
                acc_frag.x[reg_idx] = static_cast<T_OUTPUT>(acc_frag_fp32[wmma_tile_idx_m][wmma_tile_idx_n].x[reg_idx]);
                // acc_frag[wmma_tile_idx_m][wmma_tile_idx_n].x[reg_idx] += static_cast<T_OUTPUT>(acc_frag_fp32[wmma_tile_idx_m][wmma_tile_idx_n].x[reg_idx]);
            }
            __syncthreads();
            size_t M_idx = block_tile_start_m + warp_m_id * WARP_TILE_SIZE_M + wmma_tile_idx_m * WMMA_TILE_SIZE_M;
            size_t N_idx = block_tile_start_n + warp_n_id * WARP_TILE_SIZE_N + wmma_tile_idx_n * WMMA_TILE_SIZE_N;
            if (M_idx < M && N_idx < N) {
                wmma::store_matrix_sync(
                    &C[M_idx * N + N_idx], acc_frag, N, wmma::mem_row_major);
            }
        }
    }
}


template<size_t BLOCK_TILE_SIZE_M, size_t BLOCK_TILE_SIZE_N, size_t BLOCK_TILE_SIZE_K, size_t GROUP_SIZE_M>
void launch_fp8_kernel(const float8_fnuz_t* A, const float8_fnuz_t* B, const float* as, const float* bs,
                       bfloat16_t* C, size_t M, size_t N, size_t K) 
{

    constexpr size_t WARP_NUM_M{BLOCK_TILE_SIZE_M / gemmconf::WARP_TILE_SIZE_M};
    constexpr size_t WARP_NUM_N{BLOCK_TILE_SIZE_N / gemmconf::WARP_TILE_SIZE_N};
    constexpr size_t WARP_NUM{WARP_NUM_M * WARP_NUM_N};
    constexpr size_t THREAD_NUM{WARP_NUM * WARP_SIZE};

#ifdef DEBUG
    std::cout << "Launching kernel with BLOCK_TILE_SIZE_M: " << BLOCK_TILE_SIZE_M
         << ", BLOCK_TILE_SIZE_N: " << BLOCK_TILE_SIZE_N
         << ", BLOCK_TILE_SIZE_K: " << BLOCK_TILE_SIZE_K
         << ", GROUP_SIZE_M: " << GROUP_SIZE_M
         << ", WARP_NUM_M: " << WARP_NUM_M
         << ", WARP_NUM_N: " << WARP_NUM_N
         << ", THREAD_NUM: " << THREAD_NUM
         << std::endl;

    const float8_fnuz_t test[16] = {1,1,1,1,1,1,1,1, 
                              1,1,1,1,1,1,1,1};

    int4 test_int4 = *reinterpret_cast<int4 const*>(&test[0]);
    union {
        float8_fnuz_t fp8_vals[16];
        int4 int4_vals;
    } test_union;
    test_union.int4_vals = test_int4;
    for (size_t i = 0; i < 2; ++i) {
        std::cout << "test_union.fp8_vals[" << i << "]: " << static_cast<float8_fnuz_t>(test_union.fp8_vals[i]) << std::endl;
    }
#endif // DEBUG
    // static_assert(THREAD_NUM <= 1024, "THREAD_NUM must be less than or equal to 1024");

    FP8_GEMM_kernel<float8_fnuz_t, bfloat16_t, 
                    BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N, BLOCK_TILE_SIZE_K, 
                    gemmconf::WARP_TILE_SIZE_M, gemmconf::WARP_TILE_SIZE_N, 
                    gemmconf::WMMA_TILE_SIZE_M, gemmconf::WMMA_TILE_SIZE_N, gemmconf::WMMA_TILE_SIZE_K, 
                    GROUP_SIZE_M>
        <<<dim3((N + BLOCK_TILE_SIZE_N - 1) / BLOCK_TILE_SIZE_N, (M + BLOCK_TILE_SIZE_M - 1) / BLOCK_TILE_SIZE_M, 1),
           dim3(THREAD_NUM), 0>>>(A, B, as, bs, C, M, N, K);
}

/*
Reference implementation of block-scale fp8 gemm
Args:
    data: Tuple that expands to:
        a: torch.Tensor[float8_e4m3fnuz] of shape [m, k], col-major
        b: torch.Tensor[float8_e4m3fnuz] of shape [n, k], col-major 
                        -> same as [k,n] row-major
        a_scale: torch.Tensor[float32] of shape [m, k // SCALE_BLOCK_SIZE], col-major
        b_scale: torch.Tensor[float32] of shape [n // SCALE_BLOCK_SIZE, k // SCALE_BLOCK_SIZE], col-major 
                        -> same as [k // SCALE_BLOCK_SIZE, n // SCALE_BLOCK_SIZE] row-major
        c: torch.Tensor[bfloat16] of shape [m, n], row-major
        SCALE_BLOCK_SIZE = 128
Returns:
    Tensor containing output in bf16
*/
template<size_t BLOCK_TILE_SIZE_M, size_t BLOCK_TILE_SIZE_N, size_t GROUP_SIZE_M>
void select_BLOCK_TILE_SIZE_K(const float8_fnuz_t* A, const float8_fnuz_t* B, const float* as, const float* bs,
                       bfloat16_t* C, size_t M, size_t N, size_t K)
{
    constexpr std::array<size_t, 3> BLOCK_TILE_SIZE_K_LIST = {128, 64, 32};

    // if (M / K > 6 && N / K > 6) {
    //     launch_fp8_kernel<BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N, 64, GROUP_SIZE_M>(A, B, as, bs, C, M, N, K);
    // } else if (BLOCK_TILE_SIZE_M <= 64 && BLOCK_TILE_SIZE_N <= 64) {
    //     launch_fp8_kernel<BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N, 128, GROUP_SIZE_M>(A, B, as, bs, C, M, N, K);
    // } else 
    if (K % BLOCK_TILE_SIZE_K_LIST[0] == 0) { // 128
        launch_fp8_kernel<BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N, BLOCK_TILE_SIZE_K_LIST[0], GROUP_SIZE_M>(A, B, as, bs, C, M, N, K);
    } else if (K % BLOCK_TILE_SIZE_K_LIST[1] == 0) { // 64
        launch_fp8_kernel<BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N, BLOCK_TILE_SIZE_K_LIST[1], GROUP_SIZE_M>(A, B, as, bs, C, M, N, K);
    } else if (K % BLOCK_TILE_SIZE_K_LIST[2] == 0) { // 32
        launch_fp8_kernel<BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N, BLOCK_TILE_SIZE_K_LIST[2], GROUP_SIZE_M>(A, B, as, bs, C, M, N, K);
    } else {
        throw std::runtime_error("K (" + std::to_string(K) + ") is not divisible by any predefined BLOCK_TILE_SIZE_K in the list.");   
    }
}


template<size_t BLOCK_TILE_SIZE_M, size_t GROUP_SIZE_M>
void select_BLOCK_TILE_SIZE_N(const float8_fnuz_t* A, const float8_fnuz_t* B, const float* as, const float* bs,
                       bfloat16_t* C, size_t M, size_t N, size_t K)
{
    constexpr std::array<size_t, 4> BLOCK_TILE_SIZE_N_LIST = {64, 32};

    // if (M / N > 6) {
    //     select_BLOCK_TILE_SIZE_K<256, 64, GROUP_SIZE_M>(A, B, as, bs, C, M, N, K);
    // } else if (N / M > 6) {
    //     select_BLOCK_TILE_SIZE_K<64, 256, GROUP_SIZE_M>(A, B, as, bs, C, M, N, K);
    // } else 
    if (N % BLOCK_TILE_SIZE_N_LIST[0] == 0) { // 256
        select_BLOCK_TILE_SIZE_K<BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N_LIST[0], GROUP_SIZE_M>(A, B, as, bs, C, M, N, K);
    } else if (N % BLOCK_TILE_SIZE_N_LIST[1] == 0) { // 128
        select_BLOCK_TILE_SIZE_K<BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N_LIST[1], GROUP_SIZE_M>(A, B, as, bs, C, M, N, K);
    // } else if (N % BLOCK_TILE_SIZE_N_LIST[2] == 0) { // 64
    //     select_BLOCK_TILE_SIZE_K<BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N_LIST[2], GROUP_SIZE_M>(A, B, as, bs, C, M, N, K);
    // } else if (N % BLOCK_TILE_SIZE_N_LIST[3] == 0) { // 32
    //     select_BLOCK_TILE_SIZE_K<BLOCK_TILE_SIZE_M, BLOCK_TILE_SIZE_N_LIST[3], GROUP_SIZE_M>(A, B, as, bs, C, M, N, K);
    } else {
        throw std::runtime_error("N (" + std::to_string(N) + ") is not divisible by any predefined BLOCK_TILE_SIZE_N in the list.");   
    }
}

void select_BLOCK_TILE_SIZE_M(const float8_fnuz_t* A, const float8_fnuz_t* B, const float* as, const float* bs,
                       bfloat16_t* C, size_t M, size_t N, size_t K)
{
    constexpr std::array<size_t, 4> BLOCK_TILE_SIZE_M_LIST = {128, 64, 32};
    
    if (M % BLOCK_TILE_SIZE_M_LIST[0] == 0) { // 128
        select_BLOCK_TILE_SIZE_N<BLOCK_TILE_SIZE_M_LIST[0], 1>(A, B, as, bs, C, M, N, K);
    } else if (M % BLOCK_TILE_SIZE_M_LIST[1] == 0) { // 64
        select_BLOCK_TILE_SIZE_N<BLOCK_TILE_SIZE_M_LIST[1], 1>(A, B, as, bs, C, M, N, K);
    } else if (M % BLOCK_TILE_SIZE_M_LIST[2] == 0) { // 32
        select_BLOCK_TILE_SIZE_N<BLOCK_TILE_SIZE_M_LIST[2], 1>(A, B, as, bs, C, M, N, K);
    // } else if (M % BLOCK_TILE_SIZE_M_LIST[3] == 0) { // 32
    //     select_BLOCK_TILE_SIZE_N<BLOCK_TILE_SIZE_M_LIST[3], 1>(A, B, as, bs, C, M, N, K);
    } else {
        throw std::runtime_error("M (" + std::to_string(M) + ") is not divisible by any predefined BLOCK_TILE_SIZE_M in the list.");   
    }
}


void bf16_mm(torch::Tensor a, torch::Tensor b, torch::Tensor as, torch::Tensor bs, torch::Tensor c) 
{
    const size_t m = a.size(0);
    const size_t n = b.size(0);
    const size_t k = a.size(1); 

    select_BLOCK_TILE_SIZE_M(static_cast<bfloat16_t*>(a.data_ptr()), static_cast<bfloat16_t*>(b.data_ptr()), 
    as.data_ptr<float>(), bs.data_ptr<float>(), static_cast<bfloat16_t*>(c.data_ptr()), m, n, k);
}

void fp8_mm(torch::Tensor a, torch::Tensor b, torch::Tensor as, torch::Tensor bs, torch::Tensor c) 
{
    const size_t m = a.size(0);
    const size_t n = b.size(0);
    const size_t k = a.size(1); 

    select_BLOCK_TILE_SIZE_M(static_cast<float8_fnuz_t*>(a.data_ptr()), static_cast<float8_fnuz_t*>(b.data_ptr()), 
    as.data_ptr<float>(), bs.data_ptr<float>(), static_cast<bfloat16_t*>(c.data_ptr()), m, n, k);
}

}
}
}
