
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hipblas.h>

template <typename typeIn, typename typeOut>
void cublas_gemm_test(
    typeIn* A, size_t ldA,
    typeIn* B, size_t ldB,
    typeOut* C, size_t ldC,
    size_t M, size_t N, size_t K);


template<>
void cublas_gemm_test<__hip_bfloat16, __hip_bfloat16>(
    __hip_bfloat16* A, size_t ldA,
    __hip_bfloat16* B, size_t ldB,
    __hip_bfloat16* C, size_t ldC,
    size_t M, size_t N, size_t K) {

  hipblasHandle_t handle;
  hipblasStatus_t status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "cuBLAS create failed: " << status << std::endl;
    return;
  }

  const float alpha = 1.0f;
  const float beta = 0.0f;

  status = hipblasGemmEx(handle,
               HIPBLAS_OP_N, HIPBLAS_OP_N,
               N, M, K,
               &alpha,
               A, HIP_R_16BF, ldA,
               B, HIP_R_16BF, ldB,
               &beta,
               C, HIP_R_16BF, ldC,
               HIP_R_32F,
               HIPBLAS_GEMM_DEFAULT);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "cuBLAS GEMM (bf16->bf16) failed: " << status << std::endl;
  }

  hipblasDestroy(handle);
}

template<>
void cublas_gemm_test<__hip_bfloat16, float>(
    __hip_bfloat16* A, size_t ldA,
    __hip_bfloat16* B, size_t ldB,
    float* C, size_t ldC,
    size_t M, size_t N, size_t K) {

  hipblasHandle_t handle;
  hipblasStatus_t status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "cuBLAS create failed: " << status << std::endl;
    return;
  }

  const float alpha = 1.0f;
  const float beta = 0.0f;

  status = hipblasGemmEx(handle,
               HIPBLAS_OP_N, HIPBLAS_OP_N,
               N, M, K,
               &alpha,
               B, HIP_R_16BF, ldB,
               A, HIP_R_16BF, ldA,
               &beta,
               C, HIP_R_32F, ldC,
               HIP_R_32F,
               HIPBLAS_GEMM_DEFAULT);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "ldA=" << ldA << ", ldB=" << ldB << ", ldC=" << ldC << std::endl;
    std::cerr << "cuBLAS GEMM (bf16->float) failed: " << status << std::endl;
  }

  hipblasDestroy(handle);
}



template<>
void cublas_gemm_test<int8_t, int8_t>(
    int8_t* A, size_t ldA,
    int8_t* B, size_t ldB,
    int8_t* C, size_t ldC,
    size_t M, size_t N, size_t K) {

  hipblasHandle_t handle;
  hipblasStatus_t status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "cuBLAS create failed: " << status << std::endl;
    return;
  }

  const float alpha = 1.0f;
  const float beta = 0.0f;

  status = hipblasGemmEx(handle,
               HIPBLAS_OP_N, HIPBLAS_OP_N,
               N, M, K,
               &alpha,
               A, HIP_R_8I, ldA,
               B, HIP_R_8I, ldB,
               &beta,
               C, HIP_R_8I, ldC,
               HIP_R_32F,
               HIPBLAS_GEMM_DEFAULT);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "cuBLAS GEMM (int8) failed: " << status << std::endl;
  }

  hipblasDestroy(handle);
}